#include <glad/glad.h>
#include <GLFW/glfw3.h>
#include <glm/glm.hpp>
#include <glm/gtc/matrix_transform.hpp>
#include <glm/gtc/type_ptr.hpp>
#include <iostream>
#include "include/n_body_algo.h"
 
// Function prototypes
void key_callback(GLFWwindow* window, int key, int scancode, int action, int mode);

// Window dimensions
const GLuint WIDTH = 800, HEIGHT = 800;


// The MAIN function, from here we start the application and run the game loop
GLFWwindow* init(){
     // Init GLFW
    glfwInit();
    // Set all the required options for GLFW
    glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 3);
    glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 3);
    glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);
    glfwWindowHint(GLFW_RESIZABLE, GL_FALSE);

    
    // Create a GLFWwindow object that we can use for GLFW's functions
    GLFWwindow* window = glfwCreateWindow(WIDTH, HEIGHT, "LearnOpenGL", nullptr, nullptr);
    glfwMakeContextCurrent(window);

    if (!gladLoadGLLoader((GLADloadproc)glfwGetProcAddress))
    {
        std::cout << "Failed to initialize GLAD" << std::endl;
        return NULL;
    }
    // Set the required callback functions
    glfwSetKeyCallback(window, key_callback);

    
    // Define the viewport dimensions
    int width, height;
    glfwGetFramebufferSize(window, &width, &height);  
    glViewport(0, 0, width, height);

    return window;
}
int main()
{
   
    GLFWwindow* window = init();

    n_body_algo n("input/3.txt");
    
    
    // Game loop
    while (!glfwWindowShouldClose(window))
    {
        // Check if any events have been activiated (key pressed, mouse moved etc.) and call corresponding response functions
        glfwPollEvents();

        // Render
        // Clear the colorbuffer
        glClearColor(0.2f, 0.3f, 0.3f, 1.0f);
        glClear(GL_COLOR_BUFFER_BIT);

        // c.draw_circle();
        n.next_timestamp();

        glfwSwapBuffers(window);
        // c.move(v);
    }

    // c.clear_VAO_VBO();
    glfwTerminate();
    return 0;
}

// Is called whenever a key is pressed/released via GLFW
void key_callback(GLFWwindow* window, int key, int scancode, int action, int mode)
{
    if (key == GLFW_KEY_ESCAPE && action == GLFW_PRESS)
        glfwSetWindowShouldClose(window, GL_TRUE);
}